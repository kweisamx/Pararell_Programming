
/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2], 	/* values at time t */
       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */
float *gvalue, *goldval, *gnewval;

int size = MAXPOINTS+2;

/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}


__global__ void gpu_init_old_val(float *a, float *b, float *c, int n){
        int j=blockIdx.x*blockDim.x+threadIdx.x;
        int m=gridDim.x*blockDim.x;
        for(int k=j; k<n; k+=m){
            a[k] = b[k];
        }
        __syncthreads();
}
__global__ void gpu_update_point(float *a, float *b, float *c, int point, int nsteps){
                int p=blockIdx.x*blockDim.x+threadIdx.x;
                float aval = a[p];
                float bval = b[p];
                float cval;
                if (p < point) {
                for (int k = 0;k<nsteps;k++){
                    if ((p== 0) || (p  == point - 1))
                        cval = 0.0;
                    else
                        cval = (2.0 * bval) - aval + (0.09 * (-2.0)*bval);
                        
                        aval = bval;
                        bval = cval;
                        __syncthreads();
                    }
                }
                b[p] = bval;
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
void do_math(int i)
{
   float dtime, c, dx, tau, sqtau;

   dtime = 0.3;
   c = 1.0;
   dx = 1.0;
   tau = (c * dtime / dx);
   sqtau = tau * tau;
   newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);
}



/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
   int  j;
   float x, fac, k, tmp;

   /* Calculate initial values based on sine curve */
   fac = 2.0 * PI;
   k = 0.0; 
   tmp = tpoints - 1;
   for (j = 0; j < tpoints; j++) {
      x = k/tmp;
      values[j] = sin (fac * x);
      k = k + 1.0;
   }
   hipMemcpy(gvalue, values, size, hipMemcpyHostToDevice);
   hipMemcpy(goldval, oldval, size, hipMemcpyHostToDevice);
   hipMemcpy(gnewval, newval, size, hipMemcpyHostToDevice);
   
   /* Initialize old values array */
   gpu_init_old_val<<<30,512>>>(goldval, gvalue, gnewval, tpoints);


   printf("Updating all points for all time steps...\n");
   
   /* Update values for each time step */
      /* Update points along line for this time step */
   gpu_update_point<<<(tpoints/512 + 1),512>>>(goldval, gvalue, gnewval, tpoints, nsteps);
   hipMemcpy(values, gvalue, size, hipMemcpyDeviceToHost);

}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 0; i < tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 9)
         printf("\n");
   }
}


/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();

    // set GPU memory 
    hipMalloc((void**)&gvalue, size);
    hipMalloc((void**)&goldval, size);
    hipMalloc((void**)&gnewval, size);

	printf("Initializing points on the line...\n");
	init_line();
	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
	
	return 0;
}
