
/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2], 	/* values at time t */
       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */
float *gvalue, *goldval, *gnewval;

int size = MAXPOINTS+2;

/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}


__global__ void gpu_init_old_val(float *a, float *b, float *c, int n){
        int j=blockIdx.x*blockDim.x+threadIdx.x;
        int m=gridDim.x*blockDim.x;
        for(int k=j; k<n; k+=m){
            a[k] = b[k];
        }
        __syncthreads();
}
__global__ void gpu_update(float *a, float *b, float *c, int n){
        int j=blockIdx.x*blockDim.x+threadIdx.x;
        int m=gridDim.x*blockDim.x;
        for(int k=j; k<n; k+=m){
            a[k] = b[k];
            b[k] = c[k];
        }
        __syncthreads();
        
}
__global__ void gpu_update_point(float *a, float *b, float *c, int n){
        for(int k=0; k<n; k++){
            a[k] = b[k];
            b[k] = c[k];
        }
        __syncthreads();
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
void do_math(int i)
{
   float dtime, c, dx, tau, sqtau;

   dtime = 0.3;
   c = 1.0;
   dx = 1.0;
   tau = (c * dtime / dx);
   sqtau = tau * tau;
   newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);
}



/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
   int i, j;
   float x, fac, k, tmp;

   /* Calculate initial values based on sine curve */
   fac = 2.0 * PI;
   k = 0.0; 
   tmp = tpoints - 1;
   for (j = 1; j <= tpoints; j++) {
      x = k/tmp;
      values[j] = sin (fac * x);
      k = k + 1.0;
   } 
   hipMemcpy(gvalue, values, size, hipMemcpyHostToDevice);
   hipMemcpy(goldval, oldval, size, hipMemcpyHostToDevice);
   hipMemcpy(gnewval, newval, size, hipMemcpyHostToDevice);
   
   /* Initialize old values array */
   gpu_init_old_val<<<30,512>>>(goldval, gvalue, gnewval, tpoints);

   hipMemcpy(values, gvalue, size, hipMemcpyDeviceToHost);
   hipMemcpy(oldval, goldval, size, hipMemcpyDeviceToHost);

   printf("Updating all points for all time steps...\n");
   
   /* Update values for each time step */
   for (i = 1; i<= nsteps; i++) {
      /* Update points along line for this time step */
      for (j = 1; j <= tpoints; j++) {
         if ((j == 1) || (j  == tpoints))
            newval[j] = 0.0;
         else
            do_math(j);
      }

      /* Update old values with new values */
      
      for (j = 1; j <= tpoints; j++) {
         oldval[j] = values[j];
         values[j] = newval[j];
      }

      //gpu_update<<<1,1>>>(goldval, gvalue, newval, tpoints);
      //cudaMemcpy(newval, gnewval, size, cudaMemcpyDeviceToHost);
      //cudaMemcpy(values, gvalue, size, cudaMemcpyDeviceToHost);
   }

}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}


/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();

    // set GPU memory 
    hipMalloc((void**)&gvalue, size);
    hipMalloc((void**)&goldval, size);
    hipMalloc((void**)&gnewval, size);

	printf("Initializing points on the line...\n");
	init_line();
	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
	
	return 0;
}
